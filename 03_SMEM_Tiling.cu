#include <stdio.h>
#include <assert.h>
#include <hip/hip_runtime.h>

#define M 1024 
#define N 512 
#define P 2048 
#define TILE_WIDTH 32

#define CUDA_CHECK(err) {if (err != hipSuccess){printf("%s in %s at line %d \n", hipGetErrorString(err), __FILE__, __LINE__);exit(EXIT_FAILURE);}}



__global__ void tiled_mat_mul_kernel(float *A, float *B, float *C, int m, int n, int p){

    // Ensure that TILE_WIDTH = BLOCK_SIZE
    assert(TILE_WIDTH == blockDim.x);
    assert(TILE_WIDTH == blockDim.y);

    int by=blockIdx.y;
    int bx=blockIdx.x;

    int ty=threadIdx.y;
    int tx=threadIdx.x;

    int i=by*TILE_WIDTH + ty;
    int j=bx*TILE_WIDTH + tx;

    __shared__ float sh_A[TILE_WIDTH][TILE_WIDTH];
    __shared__ float sh_B[TILE_WIDTH][TILE_WIDTH];

    float val=0.0f;

    for(int tileId=0;tileId<ceil((float)n/TILE_WIDTH);tileId++){

        if(i<m && (tileId*TILE_WIDTH + tx)<n){
            sh_A[ty][tx]=A[i*n + tileId*TILE_WIDTH + tx];
        }
        else{
            sh_A[ty][tx]=0.0f;
        }
        if(j<p && (tileId*TILE_WIDTH + ty)<n){
            sh_B[ty][tx]=B[(tileId*TILE_WIDTH + ty)*p + j];
        }
        else{
            sh_B[ty][tx]=0.0f;
        }
        __syncthreads();


        for(int k=0;k<TILE_WIDTH;k++){
            val+=sh_A[ty][k]*sh_B[k][tx];
        }
        __syncthreads();

    }

    if(i<m && j<p){
        C[i*p + j]=val;
    }
}



void init_matrix(float *mat , int row , int col){
    for(int i=0;i<row * col ;i++){
        mat[i]=(float)rand()/RAND_MAX;
    }
}


int main()
{
    float *h_A,*h_B,*h_C;
    float *d_A,*d_B,*d_C;

    int size_A=M*N*sizeof(float);
    int size_B=N*P*sizeof(float);
    int size_C=M*P*sizeof(float);

    //allocate host memeory
    h_A=(float*)malloc(size_A);
    h_B=(float*)malloc(size_B);
    h_C=(float*)malloc(size_C);

    //intialize matrices
    init_matrix(h_A,M,N);
    init_matrix(h_B,N,P);

    //allocate device memeory
    hipMalloc(&d_A,size_A);
    hipMalloc(&d_B,size_B);
    hipMalloc(&d_C,size_C);

    //copt data from host to device
    hipMemcpy(d_A, h_A, size_A, hipMemcpyHostToDevice);
    hipMemcpy(d_B, h_B, size_B, hipMemcpyHostToDevice);

    // Kernel execution
    dim3 dim_block(TILE_WIDTH, TILE_WIDTH, 1);
    dim3 dim_grid(ceil(P/(float)(TILE_WIDTH)), ceil(M/(float)(TILE_WIDTH)), 1);
    tiled_mat_mul_kernel<<<dim_grid, dim_block>>>(d_A, d_B, d_C, M, N, P);

    hipMemcpy(h_C, d_C, size_C, hipMemcpyDeviceToHost);

    // Free memory
    hipFree(d_A);
    hipFree(d_B);
    hipFree(d_C);
}