#include <bits/stdc++.h>
#include <hip/hip_runtime.h>
using namespace std;

#define BLOCK_SIZE 16
#define N 32

#define CUDA_CHECK(err) \
    if (err != hipSuccess) { \
        printf("CUDA Error: %s at %s:%d\n", hipGetErrorString(err), __FILE__, __LINE__); \
        exit(-1); \
    }

__global__ void swizzling(float* in, float* out, int width)
{
    __shared__ float tile[BLOCK_SIZE][BLOCK_SIZE];

    int x = blockIdx.x * BLOCK_SIZE + threadIdx.x;
    int y = blockIdx.y * BLOCK_SIZE + threadIdx.y;

    if (x < width && y < width)
        tile[threadIdx.y][threadIdx.x ^ threadIdx.y] = in[y * width + x];

    __syncthreads();

    int tx = blockIdx.y * BLOCK_SIZE + threadIdx.x;
    int ty = blockIdx.x * BLOCK_SIZE + threadIdx.y;

    if (tx < width && ty < width)
        out[ty * width + tx] = tile[threadIdx.x ^ threadIdx.y][threadIdx.y];
}


void cpuTranspose(float* out, const float* in, int width)
{
    for (int i = 0; i < width; i++)
        for (int j = 0; j < width; j++)
            out[j * width + i] = in[i * width + j];
}


int main()
{
    const int size = N * N;
    const size_t bytes = size * sizeof(float);


    float *h_in = (float*)malloc(bytes);
    float *h_out = (float*)malloc(bytes);
    float *h_ref = (float*)malloc(bytes);

    for (int i = 0; i < N; ++i)
        for (int j = 0; j < N; ++j)
            h_in[i * N + j] = i * N + j;


    float *d_in, *d_out;
    CUDA_CHECK(hipMalloc(&d_in, bytes));
    CUDA_CHECK(hipMalloc(&d_out, bytes));
    CUDA_CHECK(hipMemcpy(d_in, h_in, bytes, hipMemcpyHostToDevice));


    dim3 block(BLOCK_SIZE, BLOCK_SIZE);
    dim3 grid((N + BLOCK_SIZE - 1) / BLOCK_SIZE, (N + BLOCK_SIZE - 1) / BLOCK_SIZE);

    hipEvent_t start, stop;
    CUDA_CHECK(hipEventCreate(&start));
    CUDA_CHECK(hipEventCreate(&stop));

    CUDA_CHECK(hipEventRecord(start));

    swizzling<<<grid, block>>>(d_in, d_out, N);
    CUDA_CHECK(hipGetLastError());
    CUDA_CHECK(hipDeviceSynchronize());

    CUDA_CHECK(hipEventRecord(stop));
    CUDA_CHECK(hipEventSynchronize(stop));

    float ms = 0.0f;
    CUDA_CHECK(hipEventElapsedTime(&ms, start, stop));

    CUDA_CHECK(hipMemcpy(h_out, d_out, bytes, hipMemcpyDeviceToHost));


    cpuTranspose(h_ref, h_in, N);

    bool correct = true;
    for (int i = 0; i < size; i++) {
        if (fabs(h_ref[i] - h_out[i]) > 1e-5) {
            correct = false;
            printf("Mismatch at index %d: CPU=%f GPU=%f\n", i, h_ref[i], h_out[i]);
            break;
        }
    }

    printf("\nCUDA Swizzling Transpose completed in %.4f ms\n", ms);
    printf("Matrix transpose correct: %s\n", correct ? "YES" : "NO");

    printf("\nSample output matrix (%dx%d):\n", N, N);
    for (int i = 0; i < N; ++i) {
        for (int j = 0; j < N; ++j)
            printf("%5.0f ", h_out[i * N + j]);
        printf("\n");
    }


    CUDA_CHECK(hipFree(d_in));
    CUDA_CHECK(hipFree(d_out));
    free(h_in);
    free(h_out);
    free(h_ref);

    CUDA_CHECK(hipEventDestroy(start));
    CUDA_CHECK(hipEventDestroy(stop));

    return 0;
}
