#include <bits/stdc++.h>
#include <hip/hip_runtime.h>
using namespace std;


#define BLOCK_SIZE 16
#define N 32


#define CUDA_CHECK(err) \
    if (err != hipSuccess) { \
        printf("CUDA Error: %s at %s:%d\n", hipGetErrorString(err), __FILE__, __LINE__); \
        exit(-1); \
    }


__global__ void skewedRowTranspose(float *out, const float *in, int width) {
    __shared__ float tile[BLOCK_SIZE][BLOCK_SIZE];

    int x = blockIdx.x * BLOCK_SIZE + threadIdx.x;
    int y = blockIdx.y * BLOCK_SIZE + threadIdx.y;

    if (x < width && y < width) {
        tile[threadIdx.y][(threadIdx.x + threadIdx.y) % BLOCK_SIZE] = in[y * width + x];
    }

    __syncthreads();

    int x_t = blockIdx.y * BLOCK_SIZE + threadIdx.x;
    int y_t = blockIdx.x * BLOCK_SIZE + threadIdx.y;

    if (x_t < width && y_t < width) {
        out[y_t * width + x_t] = tile[(threadIdx.x + threadIdx.y) % BLOCK_SIZE][threadIdx.y];
    }
}


int main() {
    const int size = N * N;
    const size_t bytes = size * sizeof(float);

    float *h_in = (float *)malloc(bytes);
    float *h_out = (float *)malloc(bytes);

    for (int i = 0; i < N; ++i)
        for (int j = 0; j < N; ++j)
            h_in[i * N + j] = i * N + j;

    float *d_in, *d_out;
    CUDA_CHECK(hipMalloc((void **)&d_in, bytes));
    CUDA_CHECK(hipMalloc((void **)&d_out, bytes));

    CUDA_CHECK(hipMemcpy(d_in, h_in, bytes, hipMemcpyHostToDevice));

    dim3 block(BLOCK_SIZE, BLOCK_SIZE);
    dim3 grid((N + BLOCK_SIZE - 1) / BLOCK_SIZE, (N + BLOCK_SIZE - 1) / BLOCK_SIZE);


    hipEvent_t start, stop;
    CUDA_CHECK(hipEventCreate(&start));
    CUDA_CHECK(hipEventCreate(&stop));

    CUDA_CHECK(hipEventRecord(start));

    skewedRowTranspose<<<grid, block>>>(d_out, d_in, N);
    CUDA_CHECK(hipGetLastError());
    CUDA_CHECK(hipDeviceSynchronize());

    CUDA_CHECK(hipEventRecord(stop));
    CUDA_CHECK(hipEventSynchronize(stop));

    float ms = 0.0f;
    hipEventElapsedTime(&ms, start, stop);

    CUDA_CHECK(hipMemcpy(h_out, d_out, bytes, hipMemcpyDeviceToHost));


    printf("\nGPU Skewed Transpose Completed in %.3f ms\n", ms);
    printf("Sample Output Matrix (%d x %d):\n", N, N);
    for (int i = 0; i < N; ++i) {
        for (int j = 0; j < N; ++j)
            printf("%5.0f ", h_out[i * N + j]);
        printf("\n");
    }

    free(h_in);
    free(h_out);
    hipFree(d_in);
    hipFree(d_out);
    hipEventDestroy(start);
    hipEventDestroy(stop);

    return 0;
}
