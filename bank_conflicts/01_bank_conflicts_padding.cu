  #include <hip/hip_runtime.h>
  #include <stdio.h>
  #include <stdlib.h>
  #include <time.h>

  // Dimensions
  #define N 32  // 32x32 matrix (warp size)
  #define BLOCK_SIZE 16


  __global__ void transposeWithPadding(float *out , float *in , int width){
        __shared__ float tile[BLOCK_SIZE][BLOCK_SIZE + 1]; // +1 to avoid bank conflicts
        int x=blockIdx.x*BLOCK_SIZE + threadIdx.x;
        int y=blockIdx.y*BLOCK_SIZE + threadIdx.y;
        if(x<width && y<width){
            tile[threadIdx.y][threadIdx.x]=in[y*width + x];
        }
        __syncthreads();

        //now lets transpose the matrix okay
        //here we do block swapping so we swap blockIdx.x and blockIdx.y
        x=blockIdx.y*BLOCK_SIZE + threadIdx.x; //note the swap of blockIdx.x and blockIdx.y
        y=blockIdx.x*BLOCK_SIZE + threadIdx.y;
        if(x<width && y<width){
            out[y*width + x]=tile[threadIdx.x][threadIdx.y]; //normal transpose by changing indices of x and y
        }

  }


  int main() {
      const int size = N * N;
      float *h_in = (float*)malloc(size * sizeof(float));
      float *h_out_with_pad = (float*)malloc(size * sizeof(float));

      // Initialize input matrix
      srand(time(NULL));
      for (int i = 0; i < size; i++) {
          h_in[i] = rand() / (float)RAND_MAX;
      }

      // Allocate GPU memory
      float *d_in, *d_out_with_pad;
      hipMalloc(&d_in, size * sizeof(float));
      hipMalloc(&d_out_with_pad, size * sizeof(float));

      // Copy input to GPU
      hipMemcpy(d_in, h_in, size * sizeof(float), hipMemcpyHostToDevice);

      // Grid dimensions (for 32x32 matrix)
      dim3 block(BLOCK_SIZE, BLOCK_SIZE);
      dim3 grid(1, 1);  // Single block for simplicity

      // Run without padding
      hipEvent_t start, stop;
      hipEventCreate(&start);
      hipEventCreate(&stop);


      // Run with padding
      hipEventRecord(start);
      transposeWithPadding<<<grid, block>>>(d_out_with_pad, d_in, N);
      hipEventRecord(stop);
      hipEventSynchronize(stop);
      float time_with_pad;
      hipEventElapsedTime(&time_with_pad, start, stop);

      // Copy results back
      hipMemcpy(h_out_with_pad, d_out_with_pad, size * sizeof(float), hipMemcpyDeviceToHost);

      printf("With padding time: %.3f ms (no conflicts)\n", time_with_pad);

      // Cleanup
      free(h_in);
      free(h_out_with_pad);
      hipFree(d_in);
      hipFree(d_out_with_pad);
      hipEventDestroy(start);
      hipEventDestroy(stop);

      return 0;
  }
